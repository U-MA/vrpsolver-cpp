#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

extern "C"
{
#include "vrp_io.h"
#include "vrp_types.h"
}

#include "device_simulation.h"
#include "vehicle_manager.h"


__global__
void testTransfer(vrp_problem *device_vrp, VehicleManager *device_vms,
		      thrust::device_vector<int> device_costs)
{
    if (threadIdx.x + blockIdx.x == 0)
    {
        printf("CHECK device_vrp MEMBER\n");
	    printf("\tdevice_vrp->vertnum      %d\n",
               device_vrp->vertnum);
        printf("\tdevice_vrp->demand[0]    %d\n"
               "\tdevice_vrp->demand[3]    %d\n",
               device_vrp->demand[0], device_vrp->demand[3]);
        printf("\tdevice_vrp->dist.cost[0] %d\n",
               device_vrp->dist.cost[0]);

        printf("CHECK device_vms\n");
        printf("\tdevice_vms[0].size()     %d\n",
               device_vms[0].size());
        printf("\tdevice_vms[99].siz()     %d\n",
               device_vms[99].size());
        printf("\tdevice_vms[0]'s cost     %d\n",
               device_vms[0].computeTotalCost(device_vrp));
        printf("\tdevice_vms[99]'s cost    %d\n",
               device_vms[99].computeTotalCost(device_vrp));

        printf("\tdevice_vms[0] move customer 1\n");
        device_vms[0].move(device_vrp, 1);

        printf("\tdevice_vms[0]'s cost     %d\n",
               device_vms[0].computeTotalCost(device_vrp));
        printf("\tdevice_vms[99]'s cost    %d\n",
               device_vms[99].computeTotalCost(device_vrp));
    }
}

__global__
void randomSimulation(vrp_problem *vrp, VehicleManager *device_vms,
                      thrust::device_vector<int> device_costs)
{
    __shared__ int *candidates;
    __shared__ int candidate_size;

    int bid      = blockIdx.x;
    int customer = threadIdx.x;

    while (device_vms[bid].isFinish(device_vrp))
    {
    }
}



int main(int argc, char **argv)
{
    char infile[200];
    strcpy(infile, "Vrp-All/E/E-n13-k4.vrp");
    
    vrp_problem *host_vrp = (vrp_problem *)calloc(1, sizeof(vrp_problem));
    vrp_io(host_vrp, infile);

    vrp_problem *device_vrp = NULL;
    hipMalloc((void **)&device_vrp, sizeof(vrp_problem));

    hipMemcpy(device_vrp, host_vrp, sizeof(vrp_problem),
               hipMemcpyHostToDevice);

    int    *device_cost = NULL;
    size_t device_cost_bytes = host_vrp->edgenum * sizeof(int);
    hipMalloc((void **)&device_cost, device_cost_bytes);
    hipMemcpy(device_cost, host_vrp->dist.cost, device_cost_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(&device_vrp->dist.cost, &device_cost, sizeof(int *),
               hipMemcpyHostToDevice);

    int *device_demand = NULL;
    size_t device_demand_bytes = host_vrp->vertnum * sizeof(int);
    hipMalloc((void **)&device_demand, device_demand_bytes);
    hipMemcpy(device_demand, host_vrp->demand, device_demand_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(&device_vrp->demand, &device_demand, sizeof(int *),
               hipMemcpyHostToDevice);


    VehicleManager host_vm;
    VehicleManager *device_vms;

    hipMalloc((void **)&device_vms, 100 * sizeof(VehicleManager));
    for (int i=0; i < 100; i++)
        hipMemcpy(&device_vms[i], &host_vm, sizeof(VehicleManager),
                   hipMemcpyHostToDevice);

    thrust::device_vector<int> device_vector(100);

    testTransfer<<<100, host_vrp->vertnum>>>(device_vrp, device_vms, device_vector);

    int min = thrust::reduce(device_vector.begin(), device_vector.end(), (int) 1e6,
                             thrust::minimum<int>());

    std::cout << "min cost " << min << std::endl;

    return 0;
}
