#include "device_vrp.h"

DeviceVrp::DeviceVrp(void)
{
    hipMalloc((void **)&vrp_, sizeof(vrp_problem));
}

DeviceVrp::DeviceVrp(const HostVrp& host_vrp)
{
    hipMalloc((void **)&vrp_, sizeof(vrp_problem));

    int vertnum = host_vrp.customer_size()+1;
    hipMemcpy(&vrp_->vertnum, &vertnum, sizeof(int),
               hipMemcpyHostToDevice);

    int vehicle_size = host_vrp.vehicle_size();
    hipMemcpy(&vrp_->numroutes, &vehicle_size, sizeof(int),
               hipMemcpyHostToDevice);

    int capacity = host_vrp.capacity();
    hipMemcpy(&vrp_->capacity, &capacity, sizeof(int),
               hipMemcpyHostToDevice);

    int *host_cost = new int[host_vrp->edge_size()];
    for (int i=0; i < vertnum; i++)
        for (int j=0; j < i; j++)
            host_cost[INDEX(i, j)] = host_vrp->cost(i, j);

    int *device_cost = NULL;
    size_t device_cost_bytes = host_vrp->edge_size() * sizeof(int);
    hipMalloc((void **)&device_cost, device_cost_bytes);
    hipMemcpy(device_cost, host_cost, device_cost_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(&vrp_->dist.cost, &device_cost, sizeof(int *),
               hipMemcpyHostToDevice);

    delete[] host_cost;

    int *host_demand = new int[vertnum];
    for (int i=0; i < vertnum; i++)
        host_demand[i] = host_vrp->demand(i);

    int *device_demand = NULL;
    size_t device_demand_bytes = vertnum * sizeof(int);
    hipMalloc((void **)&device_demand, device_demand_bytes);
    hipMemcpy(device_demand, host_demand, device_demand_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(&vrp_->demand, &device_demand, sizeof(int *),
               hipMemcpyHostToDevice);

    delete[] host_demand;
}

DeviceVrp::~DeviceVrp(void)
{
    /* TODO */
}

const DeviceVrp& operator=(const HostVrp& host_vrp)
{
    /* TODO */
}
