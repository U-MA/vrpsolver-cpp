extern "C"
{
#include "vrp_types.h"
}

#include "device_vrp.h"

vrp_problem *createVrpOnDevice(void)
{
    vrp_problem *device_vrp = NULL;
    hipMalloc((void **)&device_vrp, sizeof(vrp_problem));
    return device_vrp;
}

void deleteVrpOnDevice(vrp_problem *device_vrp)
{
    hipFree(device_vrp->demand);
    hipFree(device_vrp->dist.cost);
    hipFree(device_vrp);
}

static void copyHostToDevice(int **device, int *host, size_t size_bytes);

void copyVrpHostToDevice(vrp_problem *device_vrp, const vrp_problem *host_vrp)
{
    hipMemcpy(device_vrp, host_vrp, sizeof(vrp_problem),
               hipMemcpyHostToDevice);

    copyHostToDevice(&device_vrp->dist.cost, host_vrp->dist.cost,
                         host_vrp->edgenum * sizeof(int));
    copyHostToDevice(&device_vrp->demand,    host_vrp->demand,
                         host_vrp->vertnum * sizeof(int));
}


static void copyHostToDevice(int **device_member, int *host_member, size_t size_bytes)
{
    int *device_ptr = NULL;
    hipMalloc((void **)&device_ptr, size_bytes);
    hipMemcpy(device_ptr, host_member, size_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(device_member, &device_ptr, sizeof(int *),
               hipMemcpyHostToDevice);
}
