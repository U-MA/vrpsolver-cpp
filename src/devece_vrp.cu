extern "C"
{
#include "vrp_types.h"
}

#include "device_vrp.h"

vrp_problem *createVrpOnDevice(void)
{
    vrp_problem *device_vrp = NULL;
    hipMalloc((void **)&device_vrp, sizeof(vrp_problem));
    return device_vrp;
}

static void transferHostToDevice(int **device_member, int *host_member, size_t size_bytes)
{
    int *device_ptr = NULL;
    hipMalloc((void **)&device_ptr, size_bytes);
    hipMemcpy(device_ptr, host_member, size_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(device_member, &device_ptr, sizeof(int *),
               hipMemcpyHostToDevice);
}

void transferVrpHostToDevice(vrp_problem *device_vrp, const vrp_problem *host_vrp)
{
    hipMemcpy(device_vrp, host_vrp, sizeof(vrp_problem),
               hipMemcpyHostToDevice);

    transferHostToDevice(&device_vrp->dist.cost, host_vrp->dist.cost,
                         host_vrp->edgenum * sizeof(int));
    transferHostToDevice(&device_vrp->demand,    host_vrp->demand,
                         host_vrp->vertnum * sizeof(int));
}
